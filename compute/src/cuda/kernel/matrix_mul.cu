
#include <hip/hip_runtime.h>
#include <cstdint>
#include <float.h>
#include <stdio.h>

# define TILE_SIZE 16

__global__ void matrix_multiply_shared(
    const double *A, const double *B, double *C, int M, int N, int K)
{
    __shared__ double Asub[TILE_SIZE][TILE_SIZE];
    __shared__ double Bsub[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    double sum = 0.0;

    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; ++t)
    {
        if (row < M && t * TILE_SIZE + threadIdx.x < N)
            Asub[threadIdx.y][threadIdx.x] = A[row * N + t * TILE_SIZE + threadIdx.x];
        else
            Asub[threadIdx.y][threadIdx.x] = 0.0;

        if (col < K && t * TILE_SIZE + threadIdx.y < N)
            Bsub[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * K + col];
        else
            Bsub[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();

        for (int i = 0; i < TILE_SIZE; ++i)
            sum += Asub[threadIdx.y][i] * Bsub[i][threadIdx.x];

        __syncthreads();
    }

    if (row < M && col < K)
        C[row * K + col] = sum;
}


extern "C" void launch_matrix_multiply(const double *hA, const double *hB, double *hC, int M, int N, int K)
{
    // 1. Size bookkeeping
    size_t sizeA = static_cast<size_t>(M) * N * sizeof(double);
    size_t sizeB = static_cast<size_t>(N) * K * sizeof(double);
    size_t sizeC = static_cast<size_t>(M) * K * sizeof(double);

    // 2. Device buffers
    double *dA, *dB, *dC;
    hipMalloc(&dA, sizeA);
    hipMalloc(&dB, sizeB);
    hipMalloc(&dC, sizeC);

    // 3. Copy inputs
    hipMemcpy(dA, hA, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, sizeB, hipMemcpyHostToDevice);

    // 4. Launch
    dim3 block(16, 16);
    dim3 grid((K + block.x - 1) / block.x,
              (M + block.y - 1) / block.y);

    matrix_multiply_shared<<<grid, block>>>(dA, dB, dC, M, N, K);

    // 5. Error checks
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        fprintf(stderr, "Launch failed: %s\n", hipGetErrorString(err));

    err = hipDeviceSynchronize();
    if (err != hipSuccess)
        fprintf(stderr, "Kernel error: %s\n", hipGetErrorString(err));

    // 6. Copy result back
    hipMemcpy(hC, dC, sizeC, hipMemcpyDeviceToHost);

    // 7. Cleanup
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
}
